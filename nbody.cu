#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "vector.h"
#include "config.h"
#include "planets.h"
#include "compute.h"


// represents the objects in the system.  Global variables
vector3 *hVel, *d_hVel;
vector3 *hPos, *d_hPos;
double *mass;
vector3* values;
vector3** accels;
vector3* hVel_d;
vector3* hPos_d;
double* mass_d;
vector3* values_d;
vector3** accels_d;


//initHostMemory: Create storage for numObjects entities in our system
//Parameters: numObjects: number of objects to allocate
//Returns: None
//Side Effects: Allocates memory in the hVel, hPos, and mass global variables
void initHostMemory(int numObjects)
{
	hVel = (vector3 *)malloc(sizeof(vector3) * numObjects);
	hPos = (vector3 *)malloc(sizeof(vector3) * numObjects);
	mass = (double *)malloc(sizeof(double) * numObjects);
	
	values = (vector3*)malloc(sizeof(vector3) * numObjects * numObjects);
	accels = (vector3**)malloc(sizeof(vector3*) * numObjects);
	for (int i = 0; i < numObjects; i++) {
		accels[i] = &values[i * numObjects];
	}
	
	hipMalloc((void**)&hVel_d, (sizeof(vector3) * numObjects));	
	hipMalloc((void**)&hPos_d, (sizeof(vector3) * numObjects));	
	hipMalloc((void**)&mass_d, (sizeof(double) * numObjects));

	hipMalloc((void**)&values_d, (sizeof(vector3) * numObjects * numObjects));
	hipMalloc((void**)&accels_d, (sizeof(vector3*) * numObjects));	
}

//freeHostMemory: Free storage allocated by a previous call to initHostMemory
//Parameters: None
//Returns: None
//Side Effects: Frees the memory allocated to global variables hVel, hPos, and mass.
void freeHostMemory()
{
	free(hVel);
	free(hPos);
	free(mass);
	
	free(values);
	free(accels);

	hipFree(hVel_d);
	hipFree(hPos_d);
	hipFree(mass_d);
	
	hipFree(values_d);
	hipFree(accels_d);
}


//planetFill: Fill the first NUMPLANETS+1 entries of the entity arrays with an estimation
//				of our solar system (Sun+NUMPLANETS)
//Parameters: None
//Returns: None
//Fills the first 8 entries of our system with an estimation of the sun plus our 8 planets.
void planetFill(){
	int i,j;
	double data[][7]={SUN,MERCURY,VENUS,EARTH,MARS,JUPITER,SATURN,URANUS,NEPTUNE};
	for (i=0;i<=NUMPLANETS;i++){
		for (j=0;j<3;j++){
			hPos[i][j]=data[i][j];
			hVel[i][j]=data[i][j+3];
		}
		mass[i]=data[i][6];
	}
}

//randomFill: FIll the rest of the objects in the system randomly starting at some entry in the list
//Parameters: 	start: The index of the first open entry in our system (after planetFill).
//				count: The number of random objects to put into our system
//Returns: None
//Side Effects: Fills count entries in our system starting at index start (0 based)
void randomFill(int start, int count)
{
	int i, j, c = start;
	for (i = start; i < start + count; i++)
	{
		for (j = 0; j < 3; j++)
		{
			hVel[i][j] = (double)rand() / RAND_MAX * MAX_DISTANCE * 2 - MAX_DISTANCE;
			hPos[i][j] = (double)rand() / RAND_MAX * MAX_VELOCITY * 2 - MAX_VELOCITY;
			mass[i] = (double)rand() / RAND_MAX * MAX_MASS;
		}
	}
}

//printSystem: Prints out the entire system to the supplied file
//Parameters: 	handle: A handle to an open file with write access to prnt the data to
//Returns: 		none
//Side Effects: Modifies the file handle by writing to it.
void printSystem(FILE* handle){
	int i,j;
	for (i=0;i<NUMENTITIES;i++){
		fprintf(handle,"pos=(");
		for (j=0;j<3;j++){
			fprintf(handle,"%lf,",hPos[i][j]);
		}
		printf("),v=(");
		for (j=0;j<3;j++){
			fprintf(handle,"%lf,",hVel[i][j]);
		}
		fprintf(handle,"),m=%lf\n",mass[i]);
	}
}

__global__ void cuda_test(int* deviceArray) {
	int thread_x = threadIdx.x;
	printf("thread %d working\n", thread_x);
	deviceArray[thread_x] = deviceArray[thread_x] * 2;
	printf("thread %d placed %d in deviceArray\n", thread_x, deviceArray[thread_x]);
}

int main(int argc, char **argv)
{
	clock_t t0=clock();
	int t_now;
	//srand(time(NULL));
	srand(1234);
	initHostMemory(NUMENTITIES);
	planetFill();
	randomFill(NUMPLANETS + 1, NUMASTEROIDS);
	//now we have a system.
	#ifdef DEBUG
	//printSystem(stdout);
	#endif
	for (t_now=0;t_now<DURATION;t_now+=INTERVAL){
		compute();
	}


/*
	int one = 1;
	dim3 dimGrid, dimBlock;
	dimGrid.x = 1;
	dimGrid.y = 1;
	dimGrid.z = 1;
	dimBlock.x = 32;
	dimBlock.y = 32;
	dimBlock.z = 1;
	//cuda_compute<<<dimGrid, dimBlock>>>(4);	
	
	printf("start test\n");
	int* h_arr;
	int* d_arr;

	h_arr = (int*)malloc(20 * sizeof(int));
	for (int i=0; i < 20; i++) {
		h_arr[i] = i;
	}
	
	hipMalloc((void**)&d_arr, 20 * sizeof(int));
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed : %s\n", hipGetErrorString(cudaStatus));
	}

	hipMemcpy(d_arr, h_arr, 20 * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed : %s\n", hipGetErrorString(cudaStatus));
	}

	cuda_compute<<<1,20>>>(d_arr, 4);
	//nothing_test();

	hipMemcpy(h_arr, d_arr, 20 * sizeof(int), hipMemcpyDeviceToHost);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed : %s\n", hipGetErrorString(cudaStatus));
	}

	for (int i = 0; i < 20; i++) {
		printf("Post kernal value at h_arr[%d] is %d\n", i, h_arr[i]);
	}

	free(h_arr);
	hipFree(d_arr);

	printf("done test\n");		
	return 0;
*/

/*
	size_t size_c = 256 * sizeof(int);
	int size = 256 * sizeof(int);
	//int *hostArray = new int[10];
	int* hostArray = (int*)malloc(2 * size);

	int* deviceArray;
	hipMalloc((void**)&deviceArray, size_c);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed : %s\n", hipGetErrorString(cudaStatus));
	}


	hostArray[0] = 1;
	printf("~~~Host has %d at 0~~~\n", hostArray[0]);	
	cuda_test<<<1, 256>>>(deviceArray);
	hipDeviceSynchronize();
	
	printf("before memcpy\n");
	hipMemcpy(hostArray, deviceArray, size_c, hipMemcpyDeviceToHost);
	printf("after memcpy\n");
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed : %s\n", hipGetErrorString(cudaStatus));
	}
	printf("~~~Host recived %d at 0~~~\n", hostArray[0]);	
*/

	clock_t t1=clock()-t0;
#ifdef DEBUG
	//printSystem(stdout);
#endif
	printf("This took a total time of %f seconds\n",(double)t1/CLOCKS_PER_SEC);

	freeHostMemory();
}


